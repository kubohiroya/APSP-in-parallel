#include "hip/hip_runtime.h"
#include "johnson_float.hpp"

#define THREADS_PER_BLOCK 32

__constant__ graph_cuda_t_float graph_const;

__forceinline__
__device__ int min_distance_float(float *dist, char *visited, int n) {
  float min = FLT_INF;
  int min_index = 0;
  for (int v = 0; v < n; v++) {
    if (!visited[v] && dist[v] <= min) {
      min = dist[v];
      min_index = v;
    }
  }
  return min_index;
}

__global__ void dijkstra_kernel_float(float *distanceMatrix, int *successorMatrix, char *visited_global) {
  int s = blockIdx.x * blockDim.x + threadIdx.x;
  int V = graph_const.V;

  if (s >= V) return;

  int *starts = graph_const.starts;
  float *weights = graph_const.weights;
  edge_t_float *edge_array = graph_const.edge_array;

  float *dist = &distanceMatrix[s * V];
  char *visited = &visited_global[s * V];
  for (int i = 0; i < V; i++) {
    dist[i] = FLT_INF;
    visited[i] = 0;
  }
  dist[s] = 0;
  for (int count = 0; count < V - 1; count++) {
    int u = min_distance_float(dist, visited, V);
    int u_start = starts[u];
    int u_end = starts[u + 1];
    float dist_u = dist[u];
    visited[u] = 1;
    for (int v_i = u_start; v_i < u_end; v_i++) {
      int v = edge_array[v_i].v;
      if (!visited[v] && dist_u != FLT_INF && dist_u + weights[v_i] < dist[v])
        dist[v] = dist_u + weights[v_i];
      successorMatrix[count] = 0; // FIXME
    }
  }
}

__global__ void bellman_ford_kernel_float(float *dist) {
  int E = graph_const.E;
  int e = threadIdx.x + blockDim.x * blockIdx.x;

  if (e >= E) return;
  float *weights = graph_const.weights;
  edge_t_float *edges = graph_const.edge_array;
  int u = edges[e].u;
  int v = edges[e].v;
  float new_dist = weights[e] + dist[u];
  // Make ATOMIC
  if (dist[u] != FLT_INF && new_dist < dist[v])
    atomicExch(&dist[v], new_dist); // Needs to have conditional be atomic too
}

__host__ bool bellman_ford_cuda_float(graph_cuda_t_float *gr, float *dist, int s) {
  int V = gr->V;
  int E = gr->E;
  edge_t_float *edges = gr->edge_array;
  float *weights = gr->weights;

#ifdef _OPENMP
#pragma omp parallel for
#endif
  for (int i = 0; i < V; i++) {
    dist[i] = FLT_INF;
  }
  dist[s] = 0;

  float *device_dist;
  hipMalloc(&device_dist, sizeof(float) * V);
  hipMemcpy(device_dist, dist, sizeof(float) * V, hipMemcpyHostToDevice);

  int blocks = (E + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  for (int i = 1; i <= V - 1; i++) {
    bellman_ford_kernel_float<<<blocks, THREADS_PER_BLOCK>>>(device_dist);
    hipDeviceSynchronize();
  }

  hipMemcpy(dist, device_dist, sizeof(float) * V, hipMemcpyDeviceToHost);
  bool no_neg_cycle = true;

  // use OMP to parallelize. Not worth sending to GPU
#ifdef _OPENMP
#pragma omp parallel for
#endif
  for (int i = 0; i < E; i++) {
    int u = edges[i].u;
    int v = edges[i].v;
    float weight = weights[i];
    if (dist[u] != FLT_INF && dist[u] + weight < dist[v])
      no_neg_cycle = false;
  }

  hipFree(device_dist);

  return no_neg_cycle;
}

/**************************************************************************
                        Johnson's Algorithm CUDA
**************************************************************************/

__host__ void johnson_cuda_float(graph_cuda_t_float *gr, float *distanceMatrix, int *successorMatrix) {

  //hipDeviceSetCacheConfig(hipFuncCachePreferL1);

  int deviceCount;
  hipGetDeviceCount(&deviceCount);

  for (int i = 0; i < deviceCount; i++) {
    hipDeviceProp_t deviceProps;
    hipGetDeviceProperties(&deviceProps, i);

    std::cout << "Device " << i << ": " << deviceProps.name << "\n"
              << "\tSMs: " << deviceProps.multiProcessorCount << "\n"
              << "\tGlobal mem: " << static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024 * 1024)
              << "GB \n"
              << "\tCUDA Cap: " << deviceProps.major << "." << deviceProps.minor << "\n";
  }

  // Const Graph Initialization
  int V = gr->V;
  int E = gr->E;
  // Structure of the graph
  edge_t_float *device_edge_array;
  float *device_weights;
  float *device_distanceMatrix;
  int *device_successorMatrix;
  int *device_starts;
  // Needed to run dijkstra
  char *device_visited;
  // Allocating memory
  hipMalloc(&device_edge_array, sizeof(edge_t_float) * E);
  hipMalloc(&device_weights, sizeof(float) * E);
  hipMalloc(&device_distanceMatrix, sizeof(float) * V * V);
  hipMalloc(&device_successorMatrix, sizeof(int) * V * V);
  hipMalloc(&device_visited, sizeof(char) * V * V);
  hipMalloc(&device_starts, sizeof(int) * (V + 1));

  hipMemcpy(device_edge_array, gr->edge_array, sizeof(edge_t_float) * E,
             hipMemcpyHostToDevice);
  hipMemcpy(device_weights, gr->weights, sizeof(float) * E, hipMemcpyHostToDevice);
  hipMemcpy(device_starts, gr->starts, sizeof(int) * (V + 1), hipMemcpyHostToDevice);

  graph_cuda_t_float graph_params;
  graph_params.V = V;
  graph_params.E = E;
  graph_params.starts = device_starts;
  graph_params.weights = device_weights;
  graph_params.edge_array = device_edge_array;
  // Constant memory parameters
  hipMemcpyToSymbol(HIP_SYMBOL(graph_const), &graph_params, sizeof(graph_cuda_t_float));
  // End initialization

  graph_cuda_t_float *bf_graph = new graph_cuda_t_float;
  bf_graph->V = V + 1;
  bf_graph->E = gr->E + V;
  bf_graph->edge_array = new edge_t_float[bf_graph->E];
  bf_graph->weights = new float[bf_graph->E];

  std::memcpy(bf_graph->edge_array, gr->edge_array, gr->E * sizeof(edge_t_float));
  std::memcpy(bf_graph->weights, gr->weights, gr->E * sizeof(float));
  std::memset(&bf_graph->weights[gr->E], 0, V * sizeof(float));

  float *h = new float[bf_graph->V];
  bool r = bellman_ford_cuda_float(bf_graph, h, V);
  if (!r) {
    std::cerr << "\nNegative Cycles Detected! Terminating Early\n";
    exit(1);
  }

#ifdef _OPENMP
#pragma omp parallel for
#endif
  for (int e = 0; e < E; e++) {
    int u = gr->edge_array[e].u;
    int v = gr->edge_array[e].v;
    gr->weights[e] = gr->weights[e] + h[u] - h[v];
  }

  int blocks = (V + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

  hipMemcpy(device_weights, gr->weights, sizeof(float) * E, hipMemcpyHostToDevice);

  dijkstra_kernel_float<<<blocks, THREADS_PER_BLOCK>>>(device_distanceMatrix, device_successorMatrix, device_visited);

  hipMemcpy(distanceMatrix, device_distanceMatrix, sizeof(float) * V * V, hipMemcpyDeviceToHost);

  hipError_t errCode = hipPeekAtLastError();
  if (errCode != hipSuccess) {
    std::cerr << "WARNING: A CUDA error occured: code=" << errCode << "," <<
              hipGetErrorString(errCode) << "\n";
  }

  // Remember to reweight edges back -- for every s reweight every v
  // Could do in a kernel launch or with OMP

  hipFree(device_edge_array);
  hipFree(device_weights);
  hipFree(device_distanceMatrix);
  hipFree(device_successorMatrix);
  hipFree(device_starts);
  hipFree(device_visited);

}

