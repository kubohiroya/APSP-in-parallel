#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "johnson.hpp"

#define THREADS_PER_BLOCK 32

template<typename Number>
__forceinline__
__device__
int min_distance(Number *dist, char *visited, int n, Number inf) {
  Number min = inf;
  int min_index = 0;
  for (int v = 0; v < n; v++) {
    if (!visited[v] && dist[v] <= min) {
      min = dist[v];
      min_index = v;
    }
  }
  return min_index;
}

template<typename Number>
__global__
void dijkstra_kernel(const graph_cuda_t<Number> *gr, Number *distanceMatrix, int *successorMatrix, char *visited_global, const Number inf) {
  int s = blockIdx.x * blockDim.x + threadIdx.x;
  int V = gr->V;

  if (s >= V) return;

  int *starts = gr->starts;
  Number *weights = gr->weights;
  edge_t *edge_array = gr->edge_array;

  Number *dist = &distanceMatrix[s * V];
  char *visited = &visited_global[s * V];
  
  for (int i = 0; i < V; i++) {
    dist[i] = inf;
    visited[i] = 0;
  }
  dist[s] = 0;
  for (int count = 0; count < V - 1; count++) {
    int u = min_distance<Number>(dist, visited, V, inf);
    int u_start = starts[u];
    int u_end = starts[u + 1];
    double dist_u = dist[u];
    visited[u] = 1;
    for (int v_i = u_start; v_i < u_end; v_i++) {
      int v = edge_array[v_i].v;
      if (!visited[v] && dist_u != inf && dist_u + weights[v_i] < dist[v]) {
        dist[v] = dist_u + weights[v_i];
	// successorMatrix[v_i * v + s] = edge_array[v_i].u;
      }
    }
  }
}

template<typename Number>
__global__
void bellman_ford_kernel(const graph_cuda_t<Number> *gr, Number *dist, const Number inf) {
  int E = gr->E;
  int e = threadIdx.x + blockDim.x * blockIdx.x;

  if (e >= E) return;
  Number *weights = gr->weights;
  edge_t *edges = gr->edge_array;
  int u = edges[e].u;
  int v = edges[e].v;
  Number new_dist = weights[e] + dist[u];
  // Make ATOMIC
  if (dist[u] != inf && new_dist < dist[v])
    atomicExch((unsigned long long int *) &dist[v],
               __double_as_longlong(new_dist)); // Needs to have conditional be atomic too
}

template<typename Number>
__host__
bool bellman_ford_cuda(const graph_cuda_t<Number> *gr, Number *dist, int s) {
  int V = gr->V;
  int E = gr->E;
  edge_t *edges = gr->edge_array;
  Number *weights = gr->weights;
  static const Number inf = getInf<Number>();
  
#ifdef _OPENMP
#pragma omp parallel for
#endif
  for (int i = 0; i < V; i++) {
    dist[i] = inf;
  }
  dist[s] = 0;

  Number *device_dist;
  hipMalloc(&device_dist, sizeof(Number) * V);
  hipMemcpy(device_dist, dist, sizeof(Number) * V, hipMemcpyHostToDevice);

  int blocks = (E + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  
  for (int i = 1; i <= V - 1; i++) {
    bellman_ford_kernel<Number> <<<blocks, THREADS_PER_BLOCK>>>(gr, device_dist, inf);
    hipDeviceSynchronize();
  }

  hipMemcpy(dist, device_dist, sizeof(Number) * V, hipMemcpyDeviceToHost);
  bool no_neg_cycle = true;

  // use OMP to parallelize. Not worth sending to GPU
#ifdef _OPENMP
#pragma omp parallel for
#endif
  for (int i = 0; i < E; i++) {
    int u = edges[i].u;
    int v = edges[i].v;
    Number weight = weights[i];
    if (dist[u] != inf && dist[u] + weight < dist[v])
      no_neg_cycle = false;
  }

  hipFree(device_dist);

  return no_neg_cycle;
}

/**************************************************************************
                        Johnson's Algorithm CUDA
**************************************************************************/

template<typename Number>
__host__
void johnson_cuda(graph_cuda_t<Number> *gr, Number *distanceMatrix) {
  //hipDeviceSetCacheConfig(hipFuncCachePreferL1);
  static const Number inf = getInf<Number>();

  // Const Graph Initialization
  // graph_cuda_t<Number> * gr = new graph_cuda_t<Number>;
  int V = gr->V;
  int E = gr->E;
  // Structure of the graph
  edge_t *device_edge_array;
  Number *device_weights;
  Number *device_distanceMatrix;
  int *device_starts;
  // Needed to run dijkstra
  char *device_visited;
  // Allocating memory
  hipMalloc(&device_edge_array, sizeof(edge_t) * E);
  hipMalloc(&device_weights, sizeof(Number) * E);
  hipMalloc(&device_distanceMatrix, sizeof(Number) * V * V);

  hipMalloc(&device_visited, sizeof(char) * V * V);
  hipMalloc(&device_starts, sizeof(int) * (V + 1));

  hipMemcpy(device_edge_array, gr->edge_array, sizeof(edge_t) * E,
             hipMemcpyHostToDevice);
  hipMemcpy(device_weights, gr->weights, sizeof(double) * E, hipMemcpyHostToDevice);
  hipMemcpy(device_starts, gr->starts, sizeof(int) * (V + 1), hipMemcpyHostToDevice);

  graph_cuda_t<Number> graph_params;
  graph_params.V = V;
  graph_params.E = E;
  graph_params.starts = device_starts;
  graph_params.weights = device_weights;
  graph_params.edge_array = device_edge_array;
  // Constant memory parameters
  hipMemcpyToSymbol(HIP_SYMBOL(gr), &graph_params, sizeof(graph_cuda_t<Number>));
  // End initialization

  graph_cuda_t<Number> *bf_graph = new graph_cuda_t<Number>;
  bf_graph->V = V + 1;
  bf_graph->E = gr->E + V;
  bf_graph->edge_array = new edge_t[bf_graph->E];
  bf_graph->weights = new Number[bf_graph->E];

  std::memcpy(bf_graph->edge_array, gr->edge_array, gr->E * sizeof(edge_t));
  std::memcpy(bf_graph->weights, gr->weights, gr->E * sizeof(Number));
  std::memset(&bf_graph->weights[gr->E], 0, V * sizeof(Number));

  Number *h = new Number[bf_graph->V];
  bool r = bellman_ford_cuda<Number>(bf_graph, h, V);
  if (!r) {
    std::cerr << "\nNegative Cycles Detected! Terminating Early\n";
    exit(1);
  }

#ifdef _OPENMP
#pragma omp parallel for
#endif
  for (int e = 0; e < E; e++) {
    int u = gr->edge_array[e].u;
    int v = gr->edge_array[e].v;
    gr->weights[e] = gr->weights[e] + h[u] - h[v];
  }

  int blocks = (V + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

  hipMemcpy(device_weights, gr->weights, sizeof(Number) * E, hipMemcpyHostToDevice);

  dijkstra_kernel<Number> <<<blocks, THREADS_PER_BLOCK>>>(gr, device_distanceMatrix, nullptr, device_visited, inf);

  hipMemcpy(distanceMatrix, device_distanceMatrix, sizeof(Number) * V * V, hipMemcpyDeviceToHost);

  hipError_t errCode = hipPeekAtLastError();
  if (errCode != hipSuccess) {
    std::cerr << "WARNING: A CUDA error occured: code=" << errCode << "," <<
              hipGetErrorString(errCode) << "\n";
  }

  // Remember to reweight edges back -- for every s reweight every v
  // Could do in a kernel launch or with OMP

  hipFree(device_edge_array);
  hipFree(device_weights);
  hipFree(device_distanceMatrix);
  hipFree(device_starts);
  hipFree(device_visited);
  // hipFree(gr);
}

template<typename Number>
__host__
void johnson_successor_cuda(graph_cuda_t<Number> *gr, Number *distanceMatrix, int *successorMatrix) {

  //hipDeviceSetCacheConfig(hipFuncCachePreferL1);
  static const Number inf = getInf<Number>();

  // Const Graph Initialization
  // graph_cuda_t<Number> * gr = new graph_cuda_t<Number>;
  int V = gr->V;
  int E = gr->E;
  // Structure of the graph
  edge_t *device_edge_array;
  Number *device_weights;
  Number *device_distanceMatrix;
  int *device_successorMatrix = nullptr;
  int *device_starts;
  // Needed to run dijkstra
  char *device_visited;
  // Allocating memory
  hipMalloc(&device_edge_array, sizeof(edge_t) * E);
  hipMalloc(&device_weights, sizeof(Number) * E);
  hipMalloc(&device_distanceMatrix, sizeof(Number) * V * V);
  hipMalloc(&device_successorMatrix, sizeof(int) * V * V);

  hipMalloc(&device_visited, sizeof(char) * V * V);
  hipMalloc(&device_starts, sizeof(int) * (V + 1));

  hipMemcpy(device_edge_array, gr->edge_array, sizeof(edge_t) * E,
             hipMemcpyHostToDevice);
  hipMemcpy(device_weights, gr->weights, sizeof(double) * E, hipMemcpyHostToDevice);
  hipMemcpy(device_starts, gr->starts, sizeof(int) * (V + 1), hipMemcpyHostToDevice);

  graph_cuda_t<Number> graph_params;
  graph_params.V = V;
  graph_params.E = E;
  graph_params.starts = device_starts;
  graph_params.weights = device_weights;
  graph_params.edge_array = device_edge_array;
  // Constant memory parameters
  hipMemcpyToSymbol(HIP_SYMBOL(gr), &graph_params, sizeof(graph_cuda_t<Number>));
  // End initialization

  graph_cuda_t<Number> *bf_graph = new graph_cuda_t<Number>;
  bf_graph->V = V + 1;
  bf_graph->E = gr->E + V;
  bf_graph->edge_array = new edge_t[bf_graph->E];
  bf_graph->weights = new Number[bf_graph->E];

  std::memcpy(bf_graph->edge_array, gr->edge_array, gr->E * sizeof(edge_t));
  std::memcpy(bf_graph->weights, gr->weights, gr->E * sizeof(Number));
  std::memset(&bf_graph->weights[gr->E], 0, V * sizeof(Number));

  Number *h = new Number[bf_graph->V];
  bool r = bellman_ford_cuda<Number>(bf_graph, h, V);
  if (!r) {
    std::cerr << "\nNegative Cycles Detected! Terminating Early\n";
    exit(1);
  }

#ifdef _OPENMP
#pragma omp parallel for
#endif
  for (int e = 0; e < E; e++) {
    int u = gr->edge_array[e].u;
    int v = gr->edge_array[e].v;
    gr->weights[e] = gr->weights[e] + h[u] - h[v];
  }

  int blocks = (V + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

  hipMemcpy(device_weights, gr->weights, sizeof(Number) * E, hipMemcpyHostToDevice);

  dijkstra_kernel<Number> <<<blocks, THREADS_PER_BLOCK>>>(gr, device_distanceMatrix, device_successorMatrix, device_visited, inf);

  hipMemcpy(distanceMatrix, device_distanceMatrix, sizeof(Number) * V * V, hipMemcpyDeviceToHost);

  hipError_t errCode = hipPeekAtLastError();
  if (errCode != hipSuccess) {
    std::cerr << "WARNING: A CUDA error occured: code=" << errCode << "," <<
              hipGetErrorString(errCode) << "\n";
  }

  // Remember to reweight edges back -- for every s reweight every v
  // Could do in a kernel launch or with OMP

  hipFree(device_edge_array);
  hipFree(device_weights);
  hipFree(device_distanceMatrix);
  hipFree(device_successorMatrix);
  hipFree(device_starts);
  hipFree(device_visited);
  // hipFree(gr);
}

template __host__ void johnson_cuda<double>(graph_cuda_t<double> *gr, double *distanceMatrix);
template __host__ void johnson_cuda<float>(graph_cuda_t<float> *gr, float *distanceMatrix);
template __host__ void johnson_cuda<int>(graph_cuda_t<int> *gr, int *distanceMatrix);
template __host__ void johnson_successor_cuda<double>(graph_cuda_t<double> *gr, double *distanceMatrix, int *successorMatrix);
template __host__ void johnson_successor_cuda<float>(graph_cuda_t<float> *gr, float *distanceMatrix, int *successorMatrix);
template __host__ void johnson_successor_cuda<int>(graph_cuda_t<int> *gr, int *distanceMatrix, int *successorMatrix);

