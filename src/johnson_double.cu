#include "hip/hip_runtime.h"
#include "johnson_double.hpp"

#define THREADS_PER_BLOCK 32

__constant__ graph_cuda_t_double graph_const;

__forceinline__
__device__ int min_distance_double(double *dist, char *visited, int n) {
  double min = DBL_INF;
  int min_index = 0;
  for (int v = 0; v < n; v++) {
    if (!visited[v] && dist[v] <= min) {
      min = dist[v];
      min_index = v;
    }
  }
  return min_index;
}

__global__ void dijkstra_kernel_double(double *distanceMatrix, int *successorMatrix, char *visited_global) {
  int s = blockIdx.x * blockDim.x + threadIdx.x;
  int V = graph_const.V;

  if (s >= V) return;

  int *starts = graph_const.starts;
  double *weights = graph_const.weights;
  edge_t_double *edge_array = graph_const.edge_array;

  double *dist = &distanceMatrix[s * V];
  char *visited = &visited_global[s * V];
  for (int i = 0; i < V; i++) {
    dist[i] = DBL_INF;
    visited[i] = 0;
  }
  dist[s] = 0;
  for (int count = 0; count < V - 1; count++) {
    int u = min_distance_double(dist, visited, V);
    int u_start = starts[u];
    int u_end = starts[u + 1];
    double dist_u = dist[u];
    visited[u] = 1;
    for (int v_i = u_start; v_i < u_end; v_i++) {
      int v = edge_array[v_i].v;
      if (!visited[v] && dist_u != DBL_INF && dist_u + weights[v_i] < dist[v])
        dist[v] = dist_u + weights[v_i];
      successorMatrix[count] = 0; // FIXME
    }
  }
}

__global__ void bellman_ford_kernel_double(double *dist) {
  int E = graph_const.E;
  int e = threadIdx.x + blockDim.x * blockIdx.x;

  if (e >= E) return;
  double *weights = graph_const.weights;
  edge_t_double *edges = graph_const.edge_array;
  int u = edges[e].u;
  int v = edges[e].v;
  double new_dist = weights[e] + dist[u];
  // Make ATOMIC
  if (dist[u] != DBL_INF && new_dist < dist[v])
    atomicExch((unsigned long long int *) &dist[v],
               __double_as_longlong(new_dist)); // Needs to have conditional be atomic too
}

__host__ bool bellman_ford_cuda_double(graph_cuda_t_double *gr, double *dist, int s) {
  int V = gr->V;
  int E = gr->E;
  edge_t_double *edges = gr->edge_array;
  double *weights = gr->weights;

#ifdef _OPENMP
#pragma omp parallel for
#endif
  for (int i = 0; i < V; i++) {
    dist[i] = DBL_INF;
  }
  dist[s] = 0;

  double *device_dist;
  hipMalloc(&device_dist, sizeof(double) * V);
  hipMemcpy(device_dist, dist, sizeof(double) * V, hipMemcpyHostToDevice);

  int blocks = (E + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  for (int i = 1; i <= V - 1; i++) {
    bellman_ford_kernel_double<<<blocks, THREADS_PER_BLOCK>>>(device_dist);
    hipDeviceSynchronize();
  }

  hipMemcpy(dist, device_dist, sizeof(double) * V, hipMemcpyDeviceToHost);
  bool no_neg_cycle = true;

  // use OMP to parallelize. Not worth sending to GPU
#ifdef _OPENMP
#pragma omp parallel for
#endif
  for (int i = 0; i < E; i++) {
    int u = edges[i].u;
    int v = edges[i].v;
    double weight = weights[i];
    if (dist[u] != DBL_INF && dist[u] + weight < dist[v])
      no_neg_cycle = false;
  }

  hipFree(device_dist);

  return no_neg_cycle;
}

/**************************************************************************
                        Johnson's Algorithm CUDA
**************************************************************************/

__host__ void johnson_cuda_double(graph_cuda_t_double *gr, double *distanceMatrix, int *successorMatrix) {

  //hipDeviceSetCacheConfig(hipFuncCachePreferL1);

  int deviceCount;
  hipGetDeviceCount(&deviceCount);

  for (int i = 0; i < deviceCount; i++) {
    hipDeviceProp_t deviceProps;
    hipGetDeviceProperties(&deviceProps, i);

    std::cout << "Device " << i << ": " << deviceProps.name << "\n"
              << "\tSMs: " << deviceProps.multiProcessorCount << "\n"
              << "\tGlobal mem: " << static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024 * 1024)
              << "GB \n"
              << "\tCUDA Cap: " << deviceProps.major << "." << deviceProps.minor << "\n";
  }

  // Const Graph Initialization
  int V = gr->V;
  int E = gr->E;
  // Structure of the graph
  edge_t_double *device_edge_array;
  double *device_weights;
  double *device_distanceMatrix;
  int *device_successorMatrix;
  int *device_starts;
  // Needed to run dijkstra
  char *device_visited;
  // Allocating memory
  hipMalloc(&device_edge_array, sizeof(edge_t_double) * E);
  hipMalloc(&device_weights, sizeof(double) * E);
  hipMalloc(&device_distanceMatrix, sizeof(double) * V * V);
  hipMalloc(&device_successorMatrix, sizeof(int) * V * V);
  hipMalloc(&device_visited, sizeof(char) * V * V);
  hipMalloc(&device_starts, sizeof(int) * (V + 1));

  hipMemcpy(device_edge_array, gr->edge_array, sizeof(edge_t_double) * E,
             hipMemcpyHostToDevice);
  hipMemcpy(device_weights, gr->weights, sizeof(double) * E, hipMemcpyHostToDevice);
  hipMemcpy(device_starts, gr->starts, sizeof(int) * (V + 1), hipMemcpyHostToDevice);

  graph_cuda_t_double graph_params;
  graph_params.V = V;
  graph_params.E = E;
  graph_params.starts = device_starts;
  graph_params.weights = device_weights;
  graph_params.edge_array = device_edge_array;
  // Constant memory parameters
  hipMemcpyToSymbol(HIP_SYMBOL(graph_const), &graph_params, sizeof(graph_cuda_t_double));
  // End initialization

  graph_cuda_t_double *bf_graph = new graph_cuda_t_double;
  bf_graph->V = V + 1;
  bf_graph->E = gr->E + V;
  bf_graph->edge_array = new edge_t_double[bf_graph->E];
  bf_graph->weights = new double[bf_graph->E];

  std::memcpy(bf_graph->edge_array, gr->edge_array, gr->E * sizeof(edge_t_double));
  std::memcpy(bf_graph->weights, gr->weights, gr->E * sizeof(double));
  std::memset(&bf_graph->weights[gr->E], 0, V * sizeof(double));

  double *h = new double[bf_graph->V];
  bool r = bellman_ford_cuda_double(bf_graph, h, V);
  if (!r) {
    std::cerr << "\nNegative Cycles Detected! Terminating Early\n";
    exit(1);
  }

#ifdef _OPENMP
#pragma omp parallel for
#endif
  for (int e = 0; e < E; e++) {
    int u = gr->edge_array[e].u;
    int v = gr->edge_array[e].v;
    gr->weights[e] = gr->weights[e] + h[u] - h[v];
  }

  int blocks = (V + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

  hipMemcpy(device_weights, gr->weights, sizeof(double) * E, hipMemcpyHostToDevice);

  dijkstra_kernel_double<<<blocks, THREADS_PER_BLOCK>>>(device_distanceMatrix, device_successorMatrix, device_visited);

  hipMemcpy(distanceMatrix, device_distanceMatrix, sizeof(double) * V * V, hipMemcpyDeviceToHost);

  hipError_t errCode = hipPeekAtLastError();
  if (errCode != hipSuccess) {
    std::cerr << "WARNING: A CUDA error occured: code=" << errCode << "," <<
              hipGetErrorString(errCode) << "\n";
  }

  // Remember to reweight edges back -- for every s reweight every v
  // Could do in a kernel launch or with OMP

  hipFree(device_edge_array);
  hipFree(device_weights);
  hipFree(device_distanceMatrix);
  hipFree(device_successorMatrix);
  hipFree(device_starts);
  hipFree(device_visited);

}

